#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <errno.h>
#include <ctype.h>
#include <mkl.h>
#include <time.h>

#include <algorithm> // std::min
#include <vector>
#include <chrono>
#include <utility> // std:swap
#ifdef CUDA
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsolver.h>
#include <thrust/transform_reduce.h>
#include <thrust/functional.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/inner_product.h>
#include <thrust/reduce.h>
#endif
//#define DEBUG 1

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

#ifdef DEBUG
# define DEBUG_PRINT(x) printf x
#else
# define DEBUG_PRINT(x) do {} while (0)
#endif

// convenient timer. just put these two states around a piece of code. 
#define START_TIMER {\
		struct timespec start, end; \
		double diff; \
		clock_gettime(CLOCK_MONOTONIC, &start);	/* mark start time */ \
		
#define END_TIMER \
		clock_gettime(CLOCK_MONOTONIC, &end);	/* mark the end time */\
		diff = (end.tv_sec - start.tv_sec) + 1.0*(end.tv_nsec - start.tv_nsec)/BILLION;\
		printf("elapsed time = %.3f seconds\n",  diff);\
		}


#define BILLION 1000000000L
#define MAX_MPC_ITER 100
/* Linear SVM training using interior point method;
   Good when #features is less than 20k.
*/
int K = 50; // rank
double C = 1;
double g = 0;
int T = 0; // kernel type -- see help_msg
int CMDPARA_S = 0; // cmdline arg s -- see help_msg
char *trainfilepath = NULL;
char *modelfilepath = NULL;
char *testfilepath = NULL;
int POS, NEG; // mapping between datafile class and +1, -1 rquired by SVM.
// cublas state
hipError_t cudaStat;
hipblasStatus_t stat;
hipsolverStatus_t statusH = HIPSOLVER_STATUS_SUCCESS;

// libsvmread populates the next two main data structs.
float *LABELS = NULL;
float *INST = NULL;
long N = 0; // number of training instances
long NN = 0; // first #number of instances
long d = 0; // number of features
lapack_int *IPIV;

int flag_analysis = 0; 
int flag_tensorcore = 0;


void parsecmd(int, char *[]);
void help_msg();
void libsvmread(char *filepath, float **labels, float **inst, long *n, long *nf);
void setmat(double *mat, int n, double val);
void NewtonStep(double *Z, double *D, double *M, double C, double *a, double *X, double *S, double *Xi, double *r, double *work,
				int d);
void SMWSolve(double *Z, double *D, double *M, double *b, double *work, int d);
void mpc(double *Z, double *a, double C, double *X, double *Xi, int n, int k);
void testKerMat(double *);
void rbf_kermatmul(float *Zd1, int ldz1, float *Zd2, int ldz2, float *Yd1, float *Yd2,
		float *Ad, int lda, float *Bd, int ldb,  int m, int n, int k,
		hipblasHandle_t handle);
float gaussrand();
template<typename FT>
__global__ void vecnorm(FT *Zd, int ldz, FT *ZI, int m, int k);
template<typename FT, typename FT2>
__global__ void rbf_kergen( int m, int n, FT *buf, int ldb, FT *XI, FT *XJ, FT *XIJ, int ldxij,
							FT2 gamma, FT *YI, FT *YJ);
double LRA(float *Z, int ldz, double *U, int ldu, long n, long k);
void pgd(double *Z, double *Y, double C, double *X, long n, long d, double l1);

struct daxpy_functor 
{
	const double a;
	daxpy_functor(double _a) : a{_a} {}
	__host__ __device__
	double operator()(const double& x, const double& y) const { 
		return a * x + y;
	}
};

template<typename T, typename S>
void matcpy(int m, int n,  const char *Amajor, T* A, int lda, const char *Bmajor, S* B, int ldb )
{
	if (*Amajor == 'R' && *Bmajor == 'R') {
		for(int i=0; i<m; i++) 
			for(int j=0; j<n; j++) 
				A[i*lda+j] = B[i*ldb+j]; 
	} else if (*Amajor == 'R' && *Bmajor == 'C') {
		for(int i=0; i<m; i++) 
			for(int j=0; j<n; j++) 
				A[i*lda+j] = B[i+j*ldb]; 
	} else if (*Amajor == 'C' && *Bmajor == 'R') {
		for(int i=0; i<m; i++) 
			for(int j=0; j<n; j++) 
				A[i+j*lda] = B[i*ldb+j]; 
	} else if (*Amajor == 'C' && *Bmajor == 'C') {
		for(int i=0; i<m; i++) 
			for(int j=0; j<n; j++) 
				A[i+j*lda] = B[i+j*ldb]; 
	} else {
		printf("unsupported major: Amajor=%c Bmajor=%c", *Amajor, *Bmajor); 
	}


}

// debuging devise

void writematrix(char *filename, double *A, int m, int n, int lda)
{

	FILE *f = fopen(filename, "w");
	for( int i=0; i<m; i++ ) {
		for( int j=0; j<n; j++ ) {
			fprintf(f, "%.16e", A[i*lda+j] ); // row-major
			if( j<n-1) fprintf(f, ",");
			else fprintf(f, "\n");
		}
	}

}

void predict(double *X,  double *testlabels, double *testinst, long testN, long testd)
{
	int nSV = 0, nBSV = 0;
	for( int i=0; i<N; i++ ){
		if( X[i] > 1e-3 ) {
			nSV++;
			if( X[i] < C-1e-3 ) {
				nBSV++;
			}
		}
	}

	int *iSV = (int*) malloc(sizeof(int)*nSV);
	int *iBSV = (int*) malloc(sizeof(int)*nBSV);

	int svi = 0, bsvi = 0;
	for( int i=0; i<N; i++ ) {
		if( X[i] > 1e-3 ) {
			iSV[svi++] = i;
			if( X[i] < C-1e-3 ) {
				iBSV[bsvi++] = i;
			}
		}
	}

	// calculate w=sum alpha_i y_i x_i
	double *w = (double*) calloc(d,sizeof(double));
	for( int i=0; i<nSV; i++ ) {
		int j = iSV[i]; // index
		for( int k=0; k<testd; k++ ) {
			w[k] += X[j]*LABELS[j]*INST[j*d+k];
			//w[k] += X[j]*INST[j*d+k]; // INST unlabeled by main().
		}
	}
	// calculate b
	double b = 0;
	if (T==0) { // linear SVM
		if( nBSV > 0 ) {
			for( int i=0; i<nBSV; i++ ) {
				int j = iBSV[i];
				b += LABELS[j];
				for( int k=0; k<d; k++ ) {
					b -= w[k]*INST[j*d+k];
				}
			}
			b = b/nBSV;
		} else {
			printf("Empty boundary SV! Give up.\n");
			b = 0;
		}
		printf("intercept b=%.3e\n", b);

		long cntyes = 0;
		for( int i=0; i<testN; i++ ) {
			double f = cblas_ddot(testd, w, 1, &testinst[i*testd], 1) + b;
			if( f * testlabels[i] > 0) cntyes++;
		}
		printf("prediction accuracy %.3f (%d/%d)\n", 1.0*cntyes/testN, cntyes, testN);
	} else if (T==2) { //RBF kernel
		double acc = 0;
		std::vector<double> bs(std::min(nBSV,100), 0);
		for (int j=0; j<std::min(nBSV,100); j++) {
			int jj = iBSV[j];
			double yj = LABELS[jj];
			for (int i=0; i<nSV; i++) {
				int ii = iSV[i];
				double acc2 = 0;
				for (int l=0; l<d; l++) {
					double diff = INST[ii*d+l] - INST[jj*d+l];
					acc2 += diff * diff;
				}
				yj -= X[ii]*LABELS[ii]*exp(-g*acc2);

			}
			acc += yj;
			bs[j] = yj;
			// printf("y[%d]=%.3e\n", jj, yj);
		}
		b = acc/std::min(nBSV,100);
		double sumsq = 0;
		for( int j=0; j<bs.size(); j++ ) 
			sumsq += (bs[j]-b)*(bs[j]-b);
		printf("mean b=%.6e std b=%.6e, #samples=%d\n ", b, sqrt(sumsq/bs.size()), bs.size());

		long cntyes = 0;
		for( int jj=0; jj<testN; jj++ ) {
			// double f = cblas_ddot(testd, w, 1, &testinst[i*testd], 1) + b;
			// int jj = iBSV[j];
			double f = b;
			for( int i=0; i<nSV; i++ ) {
				int ii = iSV[i];
				double acc2 = 0;
				for (int l=0; l<d; l++) {
					double diff = INST[ii*d+l] - testinst[jj*d+l];
					acc2 += diff * diff;
				}
				acc += X[ii]*LABELS[ii]*exp(-g*acc2);
			}
			if( f * testlabels[jj] > 0) cntyes++;
		}
		
		printf("prediction accuracy %.3f (%d/%d)\n", 1.0*cntyes/testN, cntyes, testN);
	}

}

// need U (Gram matrix approx U*U') for computing b
double writemodel(char *path, double *X,  double C, double *U)
{
	int nSV = 0, nBSV = 0;
	for( int i=0; i<N; i++ ){
		if( X[i] > 1e-3 ) {
			nSV++;
			if( X[i] < C-1e-3 ) {
				nBSV++;
			}
		}
	}

	int *iSV = (int*) malloc(sizeof(int)*nSV);
	int *iBSV = (int*) malloc(sizeof(int)*nBSV);

	int svi = 0, bsvi = 0;
	for( int i=0; i<N; i++ ) {
		if( X[i] > 1e-3 ) {
			iSV[svi++] = i;
			if( X[i] < C-1e-3 ) {
				iBSV[bsvi++] = i;
			}
		}
	}
	printf("#BSV %d, #SV %d\n", nBSV, nSV);
	// calculate w=sum alpha_i y_i x_i
	double b = 0;
	if (T==0) { // linear Kernel
		double *w = (double*) calloc(d,sizeof(double));
		for( int i=0; i<nSV; i++ ) {
			int j = iSV[i]; // index
			for( int k=0; k<d; k++ ) {
				w[k] += X[j]*LABELS[j]*INST[j*d+k];
			}
		}
		// calculate b

		if( nBSV > 0 ) {
			for( int i=0; i<nBSV; i++ ) {
				int j = iBSV[i];
				b += LABELS[j];
				for( int k=0; k<d; k++ ) {
					b -= w[k]*INST[j*d+k];
				}
			}
			b = b/nBSV;
		} else {
			printf("Empty boundary SV! Give up.\n");
			b = 0;
		}
	} else if (T==2) { // RBF Kernel
        {
            double acc = 0;
            std::vector<double> bs(std::min(nBSV,50), 0);
            for (int j=0; j<std::min(nBSV,50); j++) {
                int jj = iBSV[j];
                double yj = LABELS[jj];
                for (int i=0; i<nSV; i++) {
                    int ii = iSV[i];
                    double sum = 0; 
                    for (int k=0; k<K; k++) {
                        sum += U[ii*K+k] * U[jj*K+ k];
                    }
                    yj -= X[ii] * LABELS[jj] * sum; 
                }
                acc += yj;
                bs[j] = yj;
                // printf("y[%d]=%.3e\n", jj, yj);
            }
            b = acc/std::min(nBSV,50);
            double sumsq = 0;
            for( int j=0; j<bs.size(); j++ ) 
                sumsq += (bs[j]-b)*(bs[j]-b);
            printf("approx mean b=%.6e std b=%.6e, #samples=%d\n ", b, sqrt(sumsq/bs.size()), bs.size());
        }
	}

	FILE *f = fopen(path, "w");
    if (!f) {
        fprintf(stderr,"Can't open %s\n",path);
        exit(1);
    }
	fprintf(f,"svm_type c_svc\n");
	if( T== 0 )
		fprintf(f,"kernel_type linear\n");
	else if( T == 2 ) {
		fprintf(f,"kernel_type rbf\n");
		fprintf(f,"gamma %.7f\n", g);
	}
	fprintf(f,"nr_class 2\n");
	fprintf(f,"total_sv %d\n", nSV);
	fprintf(f,"rho %f\n", -b);
	fprintf(f,"label %d %d\n", POS, NEG);
	fprintf(f,"nr_sv %d %d\n", nBSV, nSV-nBSV);
	fprintf(f,"SV\n");
	for( int i=0; i<nSV; i++ ) {
		int j = iSV[i];
		fprintf(f, "%7f ", LABELS[j]*X[j]);
		for( int k=0; k<d; k++ ) {
			if( INST[j*d+k]>0 || INST[j*d+k]<0) {
				fprintf(f, "%d:%7f ", k+1, INST[j*d+k]);
			}
		}
		fprintf(f, "\n");
	}
	fclose(f);
	free(iSV); free(iBSV);
	return b;
}
void printmatrixd(char *filename, int m, int n, float* a, int lda)
{

	FILE *f = fopen(filename, "w");
	if (f == NULL) {
		printf("fault!\n");
		return;
	}
	for (int i = 0; i < m; i++) {
		//printf("i = %d\n", i);
		for (int j = 0; j < n; j++) {
			fprintf(f, "%.6f", a[i + j*lda]);
			if (j == n - 1) fprintf(f, "\n");
			else fprintf(f, ",");
		}
	}


	fclose(f);
}

void printmatrixdd(char *filename, int m, int n, float* a, int lda)
{

	FILE *f = fopen(filename, "w");
	if (f == NULL) {
		printf("fault!\n");
		return;
	}
	for (int i = 0; i < m; i++) {
		//printf("i = %d\n", i);
		for (int j = 0; j < n; j++) {
			fprintf(f, "%.6f", a[i *lda + j]);
			if (j == n - 1) fprintf(f, "\n");
			else fprintf(f, ",");
		}
	}


	fclose(f);
}


__global__
void  getR(int m, int n, float *da, int lda, float *dr, int ldr)
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	int j = threadIdx.y + blockDim.y * blockIdx.y;
	if (i < m&&j < n)
	{
		if (i <= j)
		{
			dr[i + j*ldr] = da[i + j*lda];
		}
	}
}

__global__
void myslacpyd(int m, int n, double *da, int lda, double *db, int ldb)
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	int j = threadIdx.y + blockDim.y * blockIdx.y;
	if (i < m && j < n) {
		db[i + j*ldb] = da[i + j*lda];
	}
}


__global__
void clear_trid(char uplo, int m, int n, double *a, int lda)
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	int j = threadIdx.y + blockDim.y * blockIdx.y;
	if (i < m && j < n) {
		if (uplo == 'l') {
			if (i > j) {
				a[i + j*lda] = 0;
			}
		}
		else {
			printf("clear_tri: option %c not implemented. \n", uplo);
			assert(0);
		}
	}
}


void printmatrixDevice(char *filename, float *dA, int lda, int m, int n)
{
	float ha[m*n];
	hipMemcpy(ha, dA, sizeof(float)*m*n, hipMemcpyDeviceToHost);
	printmatrixd(filename, m, n, ha, lda);
}

//transpose a Matrix
void transpose(float *dA, int lda, int m, int n, hipblasHandle_t handle, hipStream_t stream)
{
	float *dC;
	gpuErrchk(hipMalloc(&dC, sizeof(float)*m*n));
	float alpha = 1.0;
	float beta = 0.0;
	hipblasSgeam(handle,
		HIPBLAS_OP_T, HIPBLAS_OP_T,
		m, n,
		&alpha,
		dA, lda,
		&beta,
		dA, lda,
		dC, m);
	hipMemcpyAsync(dA, dC, m * n * sizeof(float), hipMemcpyDeviceToDevice, stream);
	hipFree(dC);
}

//dA is overwrite by Q, dR is overwrite by R
void QnR(float *dA, int lda, int m, int n, hipsolverHandle_t cusolverH, hipStream_t stream)
{
	float *d_tau = NULL;
	int *devInfo = NULL;
	float *d_work = NULL;
	int lwork_geqrf = 0;
	int lwork_orgqr = 0;
	int lwork = 0;

	//int info_gpu = 0;


	//const double h_one = 1;
	//const double h_minus_one = -1;


	hipMalloc(&d_tau, sizeof(float)*n);
	hipMalloc((void**)&devInfo, sizeof(int));

	hipsolverDnSgeqrf_bufferSize(
		cusolverH,
		m,
		n,
		dA,
		lda,
		&lwork_geqrf);

	hipsolverDnSorgqr_bufferSize(
		cusolverH,
		m,
		n,
		n,
		dA,
		lda,
		d_tau,
		&lwork_orgqr);


	lwork = (lwork_geqrf > lwork_orgqr) ? lwork_geqrf : lwork_orgqr;
	hipMalloc(&d_work, sizeof(int)*lwork);

	hipsolverDnSgeqrf(
		cusolverH,
		m,
		n,
		dA,
		lda,
		d_tau,
		d_work,
		lwork,
		devInfo);


	hipsolverDnSorgqr(
		cusolverH,
		m,
		n,
		n,
		dA,
		lda,
		d_tau,
		d_work,
		lwork,
		devInfo);
	return;
}

//dA is overwrite by Q, dR is overwrite by R
void QR(float *dA, int lda, int m, int n, float *dR, int ldr, hipsolverHandle_t cusolverH, hipStream_t stream, int flag)
{
	/*
	if(flag == 0)
	printmatrixDevice("realA0.csv", dA, lda, m, n);*/
	float *d_tau = NULL;
	int *devInfo = NULL;
	float *d_work = NULL;

	int lwork_geqrf = 0;
	int lwork_orgqr = 0;
	int lwork = 0;

	//int info_gpu = 0;


	//const double h_one = 1;
	//const double h_minus_one = -1;


	hipMalloc(&d_tau, sizeof(float)*n);
	hipMalloc((void**)&devInfo, sizeof(int));

	hipsolverDnSgeqrf_bufferSize(
		cusolverH,
		m,
		n,
		dA,
		lda,
		&lwork_geqrf);

	hipsolverDnSorgqr_bufferSize(
		cusolverH,
		m,
		n,
		n,
		dA,
		lda,
		d_tau,
		&lwork_orgqr);

	lwork = (lwork_geqrf > lwork_orgqr) ? lwork_geqrf : lwork_orgqr;
	hipMalloc(&d_work, sizeof(int)*lwork);

	hipsolverDnSgeqrf(
		cusolverH,
		m,
		n,
		dA,
		lda,
		d_tau,
		d_work,
		lwork,
		devInfo);


	//copy R from A and clear tri
	dim3 grid((n + 31) / 32, (n + 31) / 32);
	dim3 block(32, 32);
	/*
	myslacpyd << <grid, block,0,stream >> > (n, n, dA, lda, dR, ldr);
	clear_trid << <grid, block,0,stream >> > ('l', n, n, dR, ldr);*/
	/*
	if(flag == 0)
	printmatrixDevice("realA.csv", dA, lda, m, n);*/
	getR << <grid, block, 0, stream >> > (m, n, dA, lda, dR, ldr);
	hipStreamSynchronize(stream);
	/*
	printf("lda = %d ldr = %d\n", lda, ldr);
	if(flag == 0)
	printmatrixDevice("realR.csv", dR, ldr, ldr, n);*/

	hipsolverDnSorgqr(
		cusolverH,
		m,
		n,
		n,
		dA,
		lda,
		d_tau,
		d_work,
		lwork,
		devInfo);
	hipFree(d_tau);
	hipFree(d_work);
	return;
}



void CAQR(float *Q, int m, int n, int lda, int em, int k)
{
	struct timespec start, end;
	float diff;
	clock_gettime(CLOCK_MONOTONIC, &start);
	int nb = m % em == 0 ? m / em : m / em + 1;//how many blocks

	float *b0, *b1;
	gpuErrchk(hipMalloc(&b0, sizeof(float)*em*k));
	gpuErrchk(hipMalloc(&b1, sizeof(float)*em*k));

	hipsolverHandle_t csHandle;
	hipsolverDnCreate(&csHandle);
	hipblasHandle_t cbHandle;
	hipblasCreate(&cbHandle);

	hipStream_t stream;
	hipStreamCreate(&stream);

	//hipHostAlloc(&Q, m * n * sizeof(double), hipHostMallocDefault);//fix the memory of Q on CPU
	//printmatrixd("Q00.csv", m, n, Q, lda);
	float *rr;//store the stack of R on GPU

	gpuErrchk(hipMalloc(&rr, sizeof(float)*nb*k*k));

	//set stream to handle 
	hipblasSetStream(cbHandle, stream);
	hipsolverSetStream(csHandle, stream);

	int nr = 0;//nr-th R

	for (int i = 0; i < m*n; i += em*k)
	{
		printf("%d-th block\n", i);
		//need to be transpose
		hipMemcpyAsync(b0, Q + i, em * k * sizeof(float), hipMemcpyHostToDevice, stream);

		//printmatrixd("Q0.csv", m, n, Q, lda);
		//if (i == 0)
		//	printmatrixDevice("b00.csv", b0, em, em, k);
		//else
		//	printmatrixDevice("b11.csv", b0, em, em, k);
		transpose(b0, k, em, k, cbHandle, stream);
		//hipStreamSynchronize(stream);
		hipStreamSynchronize(stream);
		//if(i==0)
		//	printmatrixDevice("b0.csv", b0, em, em, k);
		//else
		//	printmatrixDevice("b1.csv", b0, em, em, k);
		hipMemcpyAsync(b1, b0, em * k * sizeof(float), hipMemcpyDeviceToDevice, stream);
		QR(b1, em, em, k, rr + nr, k, csHandle, stream, i);
		hipStreamSynchronize(stream);
		/*
		if (i == 0)
		{
		printmatrixDevice("q11.csv", b1, em, em, k);
		printmatrixDevice("r11.csv", rr+nr, k, k, k);
		}
		else
		{
		printmatrixDevice("q12.csv", b1, em, em, k);
		printmatrixDevice("r12.csv", rr+nr, k, k, k);
		}*/
		transpose(rr + nr, k, k, k, cbHandle, stream);
		nr += k*k;
		hipMemcpyAsync(Q + i, b1, em*k * sizeof(float), hipMemcpyDeviceToHost, stream);
	}
	transpose(rr, k, nb*k, k, cbHandle, stream);
	hipStreamSynchronize(stream);
	//printmatrixDevice("dRb.csv", rr, nb*k, nb*k, k);
	//QnR(rr, nb*k, nb*k, k, csHandle, stream);
	float *pr;

	gpuErrchk(hipMalloc(&pr, sizeof(float)*k*k));
	QR(rr, nb*k, nb*k, k, pr, k, csHandle, stream, 1);
	hipStreamSynchronize(stream);
	//printmatrixDevice("qr.csv", pr, k, k, k);
	nr = 0;

	transpose(rr, nb*k, k, nb*k, cbHandle, stream);

	for (int i = 0; i < m*n; i += em*k)
	{
		printf("%d-th gemm\n", i);
		hipMemcpyAsync(b0, Q + i, em*k * sizeof(float), hipMemcpyHostToDevice, stream);
		hipMemcpyAsync(b1, b0, em * k * sizeof(float), hipMemcpyDeviceToDevice, stream);
		float alpha = 1.0;
		float beta = 0.0;


		transpose(rr + nr, k, k, k, cbHandle, stream);
		hipStreamSynchronize(stream);
		hipblasSgemm(cbHandle,
			HIPBLAS_OP_N, HIPBLAS_OP_N,
			em, k, k,
			&alpha,
			b1, em,
			rr + nr, k,
			&beta,
			b0, em);

		nr += k*k;
		/*if (i == 0)
		{
		printmatrixDevice("z11.csv", b0, em, em, k);
		}
		else
		{
		printmatrixDevice("z12.csv", b0, em, em, k);
		}*/
		transpose(b0, em, k, em, cbHandle, stream);
		hipMemcpyAsync(Q + i, b0, em*k * sizeof(float), hipMemcpyDeviceToHost, stream);
	}
	hipStreamSynchronize(stream);
	//printmatrixdd("Q.csv", m, n, Q, n);
	clock_gettime(CLOCK_MONOTONIC, &end);
	diff = (end.tv_sec - start.tv_sec) + 1.0*(end.tv_nsec - start.tv_nsec) / BILLION;
	//printf("MPC elapsed time = %.6f seconds\n", diff);
	hipFree(b0);
	hipFree(b1);
	hipFree(pr);
	hipFree(rr);
}

int getem(int m, int n)
{

	if (1.0*m / 1000.0*n / 1000.0 * 4 / 1000.0 < 2)
		return m;
	for (int i = 2; i <= 100; i++)
	{
		if (m % i == 0)
		{
			m = m / i;
			if (1.0*m / 1000.0*n / 1000.0 * 4 / 1000.0 < 2)
				return m;
			i--;
		}
	}
	return m;
}

void ortho(float *W, int n, int k)
{
	int em = getem(n, k);
	printf("block size is %d\n", em);
	//printmatrixdd("A.csv", n, k, W, k);
	CAQR(W, n, k, k, em, k);
	printf("-----------------------Ortho done!\n");
}

int main(int argc, char *argv[])
{
	// stat = hipblasCreate(&handle);
	struct timespec start, end;
	float diff;
    // parse commandline
	parsecmd(argc, argv);
    // read LIBSVM format file
	clock_t before = clock();
	float *labels, *inst;
	long n, nf;
	libsvmread(trainfilepath, &labels, &inst, &n, &nf);
	LABELS = labels; INST = inst; N = n; d = nf;
	
	if (flag_tensorcore)
		printf("\e[31mUsing TensorCore \e[39m\n");

	if (NN!=0) {
		printf("Truncating the input file to first %ld instances\n", NN);
		N = NN; // only use the first NN instances; check -N options.
	}
	// PROCESSING the labes!! for COVTYPE
	int pos=-42, neg=-42;
	for( int i=0; i<N; i++ ) {
		if( pos == -42) {
			pos = LABELS[i];
		} else if( neg==-42 && pos != LABELS[i] ){
			neg = LABELS[i];
			break;
		}
	}
	POS = pos; NEG = neg;

	for( int i=0; i<N; i++ ) {
		if( LABELS[i] == POS ) LABELS[i] = 1;
		else if( LABELS[i] == NEG ) LABELS[i] = -1;
		else printf("Error: LABELS[%d] %.3f\n", i, LABELS[i]);
	}
	printf("found labels: %d(+1) %d(-1)\n", pos, neg);

	clock_t difference = clock() - before;
	printf("Reading files took %.3f seconds\n", 1.0*difference  / CLOCKS_PER_SEC);

	// primal-dual solution vectors X, Xi. 
	double *X, *Xi;
	X = (double*) malloc(sizeof(double)*N);
	Xi = (double*) malloc(sizeof(double)*N);


	if( T == 0 ){ 	// Linear SVM.
        printf("Linear SVM\n");
        double *Z = new double[N*d];
        double *Y = new double[N];
        matcpy( N, d, "RowMajor", Z, d, "RowMajor", INST, d );
        matcpy( N, 1, "RowMajor", Y, 1, "RowMajor", LABELS, 1 );
		for( int i=0; i<N; i++ )
			cblas_dscal(d, Y[i], &Z[i*d], 1);

        //writematrix("/Users/pwu/ownCloud/Projects/2019June_TensorSVM/Z.csv", Z, N, d, d);
		mpc(Z, Y, C, X, Xi, N, d);

		// unlabel the Z matrix;
		for( int i=0; i<N; i++ )
			cblas_dscal(d, Y[i], &Z[i*d], 1);

		// write to the model
		writemodel(modelfilepath, X, C, NULL); // No use of U

		// prediction if test file is supplied
		if( testfilepath ) {
            printf(" prediction unsupported \n");
			//double *testlabels, *testinst;
			//long testN, testd;
			//libsvmread(testfilepath, &testlabels, &testinst, &testN, &testd);
			//if( testd != d ) {
				//printf("training #feature(%d) != testing feature (%d)\n",
					   //d, testd);
				////return 0;
			//}
			//printf("\n\nPredicting on the test file %s...\n", testfilepath);
			//printf("Number of test instances %ld, test features %ld\n", testN, testd);
			//for( int i=0; i<testN; i++ ) {
				//if( testlabels[i] == POS ) testlabels[i] = 1;
				//else if( testlabels[i] == NEG ) testlabels[i] = -1;
			//}
			//predict(X,  testlabels, testinst, testN, testd);

		}
        delete[] Z;
        delete[] Y;
	} else if ( T == 2 ) { // RBF kernel.
		printf("\e[34mRBF kernel: gamma=%.3e, C=%.3e ", g, C);
		printf("Approximation Rank K=%d\e[39m\n", K);
		double *U = (double *) malloc( sizeof(double) * N*K );
		int ldu = K;
		clock_gettime( CLOCK_MONOTONIC, &start);
		double l1 = LRA(INST, d, U, ldu,  N, K);
		clock_gettime( CLOCK_MONOTONIC, &end);
		diff = (end.tv_sec - start.tv_sec) + 1.0*(end.tv_nsec - start.tv_nsec)/BILLION;
		printf("\e[95mLRA elapsed time = %.0f seconds\e[39m\n",  diff);
		// FILE *f3 = fopen("U.csv","w");
		// for( int i=0; i<N; i++ ){
		// 	for( int j=0; j<K; j++ ){
		// 		fprintf(f3, "%.6f", U[i*ldu+j]);
		// 		if( j<K-1 ) fprintf(f3,",");
		// 		else		fprintf(f3, "\n");
		// 	}
		// }
		// fclose(f3);
		clock_gettime( CLOCK_MONOTONIC, &start);
		if (CMDPARA_S == 0) { // approx IPM
			double *a = (double*) malloc(sizeof(double) * N); 
			for(int i=0; i<N; i++) a[i] = LABELS[i];
			mpc(U, a, C, X, Xi, N, K);
			free(a);
		} else if(CMDPARA_S == 1) { // projected gradient descent
			printf("unimplemented pgd\n");
#if 0
			pgd(INST, LABELS, C, X, N, d, l1);
#endif
		}
		clock_gettime( CLOCK_MONOTONIC, &end);
		diff = (end.tv_sec - start.tv_sec) + 1.0*(end.tv_nsec - start.tv_nsec)/BILLION;
		printf("\e[95mMPC elapsed time = %.0f seconds\e[39m\n",  diff);

#ifdef DEBUG
		printf("Calculating Primal/Dual Objective...");
		using namespace std::chrono;
		auto t1 = high_resolution_clock::now();
		
		double *Zd, *Ld, *Xd, *Yd;
		hipMallocManaged( &Zd, sizeof(double)*N*d );
		hipMallocManaged( &Ld, sizeof(double)*N );
		hipMallocManaged( &Xd, sizeof(double)*N );
		hipMallocManaged( &Yd, sizeof(double)*N );
		for( int i=0; i<N; i++ ) 
			for( int j=0; j<d; j++ ) 
				Zd[i+j*N] = INST[i*d+j];
		for( int i=0; i<N; i++ ) {
			Ld[i] = LABELS[i];
			Xd[i] = X[i];
		}
		hipblasHandle_t handle;
		hipblasCreate(&handle);
		rbf_kermatmul(Zd, N, Ld, Xd, N, Yd, N, N, 1, handle);
		hipDeviceSynchronize();
		double acc = 0; 
		for( int i=0; i<N; i++ ) {
			acc += X[i]*Yd[i]/2 - X[i];
		}
		printf("Original Primal objective: %.6e\n", acc);
		hipFree(Zd); hipFree(Ld); hipFree(Xd); hipFree(Yd);
		hipblasDestroy(handle);
		auto t2 = high_resolution_clock::now();
		duration<double> time_span = duration_cast<duration<double>>(t2 - t1);
		printf(" Done in %.0f seconds.\n", time_span.count());
#endif // DEBUG
		printf("Writemodel ");
		START_TIMER
		writemodel(modelfilepath, X, C, U);
		END_TIMER

		free(U);

		if( testfilepath ) {
			printf("testfilepath=%s\n", testfilepath);
			printf("-test not implemented\n");
#if 0
			double *testlabels, *testinst;
			long testN, testd;
			libsvmread(testfilepath, &testlabels, &testinst, &testN, &testd);
			if( testd != d ) {
				printf("training #feature(%d) != testing feature (%d)\n",
					   d, testd);
				//return 0;
			}
			printf("\n\nPredicting on the test file %s...\n", testfilepath);
			printf("Number of test instances %ld, test features %ld\n", testN, testd);
			for( int i=0; i<testN; i++ ) {
				if( testlabels[i] == POS ) testlabels[i] = 1;
				else if( testlabels[i] == NEG ) testlabels[i] = -1;
			}
			predict(X,  testlabels, testinst, testN, testd);
#endif
		}
	}
	// clean up
	free(INST);
	free(LABELS);

	free(X);
	free(Xi);
	// hipblasDestroy(handle);

	return 0;
}

void help_msg()
{
	printf("Usage: tensorsvm-train [options] training_set_file [model_file] \n");
	printf("options:\n"
		"-s svm_type : set type of SVM (default 0)\n"
		"	0 -- C-SVC		(multi-class classification)\n"
		"	1 -- nu-SVC		(multi-class classification)\n"
		"	2 -- one-class SVM\n"
		"	3 -- epsilon-SVR	(regression)\n"
		"	4 -- nu-SVR		(regression)\n"
		"-t kernel_type : set type of kernel function (default 2)\n"
		"	0 -- linear: u'*v\n"
		"	1 -- polynomial: (gamma*u'*v + coef0)^degree\n"
		"	2 -- radial basis function: exp(-gamma*|u-v|^2)\n"
		"	3 -- sigmoid: tanh(gamma*u'*v + coef0)\n"
		"	4 -- precomputed kernel (kernel values in training_set_file)\n"
		"-d degree : set degree in kernel function (default 3)\n"
		"-g gamma : set gamma in kernel function (default 1/num_features)\n"
		"-r coef0 : set coef0 in kernel function (default 0)\n"
		"-c cost : set the parameter C of C-SVC, epsilon-SVR, and nu-SVR (default 1)\n"
		"-n nu : set the parameter nu of nu-SVC, one-class SVM, and nu-SVR (default 0.5)\n"
		"-p epsilon : set the epsilon in loss function of epsilon-SVR (default 0.1)\n"
		"-m cachesize : set cache memory size in MB (default 100)\n"
		"-e epsilon : set tolerance of termination criterion (default 0.001)\n"
		"-h shrinking : whether to use the shrinking heuristics, 0 or 1 (default 1)\n"
		"-b probability_estimates : whether to train a SVC or SVR model for probability estimates, 0 or 1 (default 0)\n"
		"-wi weight : set the parameter C of class i to weight*C, for C-SVC (default 1)\n"
		"-v n: n-fold cross validation mode\n"
		"-q : quiet mode (no outputs)\n"
		"-s : set the solver (default 0); only works when -t !=0\n"
	    "     0 -- approx interior point method\n"
	    "     1 -- projected gradient descent\n"
		"     2 -- primal interior barrier method\n"
		"-analysis : enable computing fnorm(K-U*U'), the error of low rank approximation\n"
		"-N n_samples: only consider the first n_samples in training\n"
	);
}

void parsecmd(int argc, char *argv[])
{
	int modelfileflag = 0;
	for (int i=1; i<argc; i++) {
		if (strcmp(argv[i], "-c") == 0) {
			i++;
			C = atof(argv[i]);
		} else if (strcmp(argv[i], "-g") == 0) {
			i++;
			g = atof(argv[i]);
		} else if (strcmp(argv[i], "-test") == 0) {
			i++;
			testfilepath = argv[i];
		} else if (strcmp(argv[i], "-t") == 0) {
			i++;
			T = atoi(argv[i]);
            if (T!=0 && T!=2) {
                printf("Error: unimplemented -t %d\n", T);
                exit(1);
            }
		} else if (strcmp(argv[i], "-N") == 0) {
			i++;
			NN = atoi(argv[i]);
		} else if (strcmp(argv[i], "-k") == 0) {
			i++;
			K = atoi(argv[i]);
		} else if (strcmp(argv[i], "-s") == 0) {
		    i++;
			CMDPARA_S = atoi(argv[i]);
		} else if (strcmp(argv[i], "-analysis") == 0) {
			// no argument to this option
			flag_analysis = 1; 
		} else if (strcmp(argv[i], "-tensorcore") == 0) {
			flag_tensorcore = 1;
		} else {
			if (!modelfileflag) {
				trainfilepath = argv[i];
				modelfileflag = 1;
			} else {
				modelfilepath = argv[i];
			}
		}
	}
	if (trainfilepath == NULL) {
		help_msg();
	}
	if (modelfilepath == NULL) {
		modelfilepath = (char*)malloc(80);
		strcpy(modelfilepath, trainfilepath);
		strcat(modelfilepath, ".model");
	}

	DEBUG_PRINT( ("C=%f,gamma=%f\ntrainfile=%s\nmodelfile=%s\ntestfile=%s\n",
		C, g, trainfilepath,modelfilepath, testfilepath) );

}


#define BUF_SIZE 1000000
#define min(x,y) (( (x) < (y) ) ? (x) : (y) )
#define max(x,y) (( (x) > (y) ) ? (x) : (y) )
void libsvmread(char *file, float **labels, float **inst, long *n, long *nf)
{
	// 1st pass: determine N, d, class
	FILE *f = fopen(file, "r");
    if (!f) {
        fprintf(stderr,"Can't open %s\n", file);
        exit(1);
    }

	char line[BUF_SIZE];
	char *endptr;

	int max_index, min_index, inst_max_index;
	size_t elements, k, i, l=0;
	max_index = 0;
	min_index = 1; // our index starts from 1
	elements = 0;

	while( fgets(line, BUF_SIZE, f)  ) {
		char *idx, *val;
		// features
		int index = 0;
		// strtol gives 0 if wrong format, and precomputed kernel has <index> start from 0
		inst_max_index = -1;
		strtok(line," \t"); // label

		while (1)
		{
			idx = strtok(NULL,":"); // index:value
			val = strtok(NULL," \t");
			if(val == NULL)
				break;

			errno = 0;
			index = (int) strtol(idx,&endptr,10);
			if(endptr == idx || errno != 0 || *endptr != '\0' || index <= inst_max_index)
			{
				printf("Wrong input format at line %lu\n",l+1);
				return;
			}
			else
				inst_max_index = index;

			min_index = min(min_index, index);
			elements++;
		}
		max_index = max(max_index, inst_max_index);
		l++;
	}
	*labels = (float*) malloc( sizeof(float)*l);
	*inst = (float*) calloc( l*max_index, sizeof(float)); // row major
	*n = l; *nf = max_index;

	// 2nd pass: populate the label and instance array.
	rewind(f);
	printf("Dataset size %ld #features %d nnz=%zd sparsity=%3f%%\n",
		l, max_index, elements, 100.0*elements / ( *n * *nf) ) ;

	k=0;
	int j;
	for(i=0;i<l;i++)
	{
		char *idx, *val, *label;
		int index;

		fgets(line, BUF_SIZE, f);

		label = strtok(line," \t\n");
		if(label == NULL)
		{
			printf("Empty line at line %lu\n",i+1);
			return;
		}
		(*labels)[i] = strtod(label,&endptr);
		if(endptr == label || *endptr != '\0')
		{
			printf("Wrong input format at line %lu\n",i+1);
			return;
		}

		// features
		while(1)
		{
			idx = strtok(NULL,":");
			val = strtok(NULL," \t");
			if(val == NULL)
				break;

			index = (int) strtol(idx, &endptr, 10) - min_index; // base 1 to base 0.

			errno = 0;
			(*inst)[i* *nf + index] = strtod(val,&endptr);
			if (endptr == val || errno != 0 || (*endptr != '\0' && !isspace(*endptr)))
			{
				printf("Wrong input format at line %lu\n",i+1);
				return;
			}
			++k;
		}
	}

	fclose(f);




	DEBUG_PRINT( ("printing the sixth row of the inst matrix...\n") );
	DEBUG_PRINT( ("%f ", (*labels)[5]) );
	for( j=0; j<*nf; j++ ) {
		if( (*inst)[5* (*nf)+j] !=0  ) {
			DEBUG_PRINT( ("%d:%.3f ", j+1, (*inst)[5*(*nf)+j]) );
		}
	}
	printf("\n");

}

// [p]rojected [g]radient [d]escent solver with Nesterov's acceleration
// (momentum). 
// X is the data matrix; row major. n*d
// L is the label matrix; n

#define MAX_PGD_ITER 100000
// #define TINIT 0.1
// Acclerated Projected Gradient Descent; see
// http://www.stat.cmu.edu/~ryantibs/convexopt/lectures/prox-grad.pdf
// for details.
// Z: row-major data matrix, n*d
// L: label, n
// l1: estimated largest eigenvalue from LRA; for computing step size
#if 0
void pgd(double *Z, double *L, double C, double *X, long n, long d, double l1)
{
	// First guess the norm of Q which determines our step size.
	// using random projection. 
	double t = 1.0/l1; // maximum step size--1/L
	printf("Projected Gradient Descent with Momentum begins... t=%.3e\n", t);
	// double *x; // the solution
	// gpuErrchk(hipMallocManaged( &x, sizeof(double)*n));
	double *Zd, *Ld, *Xd1, *Xd2,  *Vd, *Yd;
	hipMallocManaged( &Zd, sizeof(double)*n*d );
	hipMallocManaged( &Ld, sizeof(double)*n );
	hipMallocManaged( &Xd1, sizeof(double)*n );
	hipMallocManaged( &Xd2, sizeof(double)*n );
	hipMallocManaged( &Vd, sizeof(double)*n );
	hipMallocManaged( &Yd, sizeof(double)*n );
	double *G = (double*) malloc( sizeof(double)*n );

	// hipMemset( Xd, 0, sizeof(double)*n );
	// initialize Xd1, Xd2 to all 0
	for( int i=0; i<n; i++ ) {
		Ld[i] = L[i];
		Xd1[i] = 0;
		Xd2[i] = 0;
	}
	for( int i=0; i<n; i++ ) 
		for( int j=0; j<d; j++ ) 
			Zd[i+j*n] = Z[i*d+j];
	
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	void project(double *X, double *Y, double *a, long n, long d);
	using namespace std::chrono;
	duration<double> proj_time {0};
	high_resolution_clock::time_point t1, t2;
	thrust::device_ptr<double> X1(Xd1), X2(Xd2), Y(Yd), V(Vd);
	int device = -1;
	hipGetDevice(&device);
	hipMemPrefetchAsync(Zd, n*d*sizeof(double), device, NULL);
	hipMemPrefetchAsync(Ld, n*sizeof(double), device, NULL);
	for (int iter=1; iter<MAX_PGD_ITER; iter++) {
		double pobj = 0;
		if ((iter-1)%1000==0) {
			rbf_kermatmul(Zd, n, Ld, Xd1, n, Yd, n, n, 1, handle);
			// hipDeviceSynchronize();

			// for (int i=0; i<n; i++)
			// 	pobj += 0.5* Xd1[i]*Yd[i] - Xd1[i];
			pobj = 0.5 * thrust::inner_product(thrust::device, X1, X1+n, Y, 0.0f);
			pobj -= thrust::reduce( X1, X1+n );
			// printf("pobj=%.6e ", pobj);
		}
		// thrust::fill(thrust::device, V, V+n, 0);
		// thrust::transform(thrust::device, X2, X2+n, V, V,
		// 				  daxpy_functor(-(iter-2.0)/(iter+1.0)));
		// thrust::transform(thrust::device, X1, X1+n, V, V,
		// 				  daxpy_functor( (2.0*iter-1.0)/(iter+1.0) ));

		// step 1: compute the gradient: g(v) = Qv-e, stored in Yd
		t1 = high_resolution_clock::now();
		for (int i=0; i<n; i++) 
			Vd[i] = (2.0*iter-1.0)/(iter+1.0) * Xd1[i] - (iter-2.0)/(iter+1.0) * Xd2[i];
		t2 = high_resolution_clock::now();
		// if((iter-1)%1000==0) printf("V: %.3e (s)\n", duration_cast<duration<double>>(t2 - t1).count());

		t1 = high_resolution_clock::now();

		hipMemPrefetchAsync(Vd, n*sizeof(double), device, NULL);
		hipMemPrefetchAsync(Yd, n*sizeof(double), device, NULL);
		rbf_kermatmul(Zd, n, Ld, Vd, n, Yd, n, n, 1, handle);
		hipDeviceSynchronize();
		t2 = high_resolution_clock::now();
		if((iter-1)==0) printf("rbf: %.3e (s)\t", duration_cast<duration<double>>(t2 - t1).count());
		
		t1 = high_resolution_clock::now();
		for (int i=0; i<n; i++)
			Yd[i] -= 1;

		for (int i=0; i<n; i++)
			Yd[i] = Yd[i]*(-t) + Vd[i];

		t2 = high_resolution_clock::now();
		// if((iter-1)%1000==0) printf("Yd: %.3e (s)\n", duration_cast<duration<double>>(t2 - t1).count());

		// printf("Yd2[0]=%.3e\n", Yd[0]);
		// Yd = Pc(Yd);

		t1 = high_resolution_clock::now();
		project(Yd, Xd2, L, n, d);
		t2 = high_resolution_clock::now();
		proj_time = duration_cast<duration<double>>(t2 - t1);
		if((iter-1)==0) printf("  project in %.3e seconds.\n", proj_time.count());
		// printf("Yd3[0]=%.3e\n", Yd[0]);
		// step 3: update & output solver state
		// compute G(x):
		t1 = high_resolution_clock::now();
		for (int i=0; i<n; i++) {
			G[i] = (Xd1[i] -Xd2[i]) / t;
		}
		t2 = high_resolution_clock::now();
		proj_time = duration_cast<duration<double>>(t2 - t1);
		// if((iter-1)%1000==0) printf("  G[i] in %.3e seconds.\n", proj_time.count());

		if((iter-1)%1000==0) {
			double gnorm = cblas_dnrm2( n, G, 1);
			printf("iter %-6d pobj=%.6e ||G||_2=%.3e 1/2*t*||G||^2=%.3e\n", iter, pobj, gnorm, 0.5*t*gnorm*gnorm);
			if (iter>0) 
				if (0.5*t*gnorm*gnorm < 1.e-4 * fabs(pobj))
					break;
				
		}
		// for (int i=0; i<n; i++)
		// 	Xd[i] = Yd[i];
		std::swap(Xd1, Xd2);

	}
	for (int i=0; i<n; i++) {
		X[i] = Xd2[i];
	}
	hipFree(Zd);
	hipFree(Ld);		
	// hipFree(Xd1);				
	// hipFree(Xd2);
	hipFree(Yd);
	free(G);
}

template <typename Proc>
double bisection(Proc f, double left, double right)
{
	double eps = 1.e-7;
	if (f(left) == 0) return left;
	if (f(right) == 0) return right;
	if (f(left)*f(right) >= 0) {
		printf("bisection fail: f(left)=%.3e, f(right)=%.3e\n", f(left), f(right));
		return 0;
	}
	while (right-left>eps) {
		double mid = (left+right)/2;
		if ( f(mid) * f(left) > 0 ) left = mid;
		else if ( f(mid) * f(right) > 0 ) right = mid;
		else {
			printf("bisection fail: f(left)=%.3e, f(right)=%.3e\n", f(left), f(right));
			return 0;
		}
	}
	return (left+right)/2;

}
// projection onto the constraint set
// Y = P_C(X)
// C: a^Tx = 0; 0<= x <= C;
// Y and X can be the same pointer. 
void project(double *X, double *Y, double *a, long n, long d)
{
	double left, right;
	std::vector<double> tt( 2*n );
	for (int i=0; i<n; i++) {
		tt[i] = (X[i]/a[i]);
		tt[i+n] = (X[i] -C)/a[i];
	}
	right = *std::max_element(tt.begin(), tt.end());
	left = *std::min_element(tt.begin(), tt.end());
	double Clocal = C;
	auto fun = [X, a, Clocal, n](double lambda){
			double sum = 0;
			for (int i=0; i<n; i++) {
				double yy = X[i] - lambda*a[i];
				if (yy < 0) {
					sum += 0;
				} else if (yy > Clocal) {
					sum += a[i] * Clocal;
				} else {
					sum += a[i] * yy;
				}
			}
			return sum;
	};
	double lambda = bisection( fun, left, right );
	// test if the bisectio indeed solves the equation:
	// printf("project: f(%.3e)=%.6e (should be zero)\n", lambda, fun(lambda));
	for (int i=0; i<n; i++) {
		Y[i] = X[i] - lambda*a[i];
		if (Y[i] < 0) Y[i] = 0;
		else if (Y[i] > C) Y[i] = C;
	}
}
#endif


// The kernel SVM is boils down to the following convex
// quadratic programming problem (P):
// min_x 1/2 x^T*Q*x - e^T*x, subject to a^T x = 0, 0<=x<=C.
// The dual problem of the last line is (D):
// max_{y,s} -1/2 x^T*Q*x - C 1^T*\Xi, subject to
//     -Q*x + a*y + s - \xi = -e, s>=0, \xi>=0
//
//
// the central path \sigma\mu KKT condition is
//     Xs = \sigma \mu *e
//     (C-X)\xi = \sigma \mu *e
//     a^T*x = 0
//     -Q*x + a*y + s - \xi = -e
//     0<=x<=c, s>=0, \xi>=0
//
// The Newton step to solve the KKT condition has Jacobian:
// [ -Q   a   I    -I ] [ Dx ]
// [ a^T  0   0    0  ] [ Dy ]
// [ S    0   X    0  ] [ Ds ]
// [ -Xi  0   0   C-X ] [ Dxi]

// Mehrotra's algorithm for the dual problem of linear and kernel SVM.
// In linear case, Q = LABELS'*INST*INST'*LABELS
// Z is labeled; a is +1/-1 labes.
void mpc(double *Z, double *a, double C, double *X, double *Xi, int N, int d)
{
	hipblasHandle_t handle; 
	hipblasCreate(&handle); 
	struct timespec start, end;
	float diff;
	// printf("mpc: N=%d, d=%d\n", N, d);
	//double *Z = INST;
	//double *X = (double*) malloc(sizeof(double) * N);
	double y = 0;
	//double *Xi = (double*) malloc(sizeof(double) * N);
	double *S = (double*) malloc(sizeof(double) * N);

	double *r = (double*) malloc(sizeof(double) * (3*N+1) );
	double *r_aff = (double*) malloc(sizeof(double) * (3*N+1));

	double *q = (double*) malloc(sizeof(double) * d);
	double *e = (double*) malloc(sizeof(double) * N);

	setmat(e, N, 1);

	cblas_dgemv(CblasRowMajor, CblasTrans, N, d, 1.0, Z, d, e, 1, 0, q, 1); // q = Z'*e;
	cblas_dgemv(CblasRowMajor, CblasNoTrans, N, d, 1.0, Z, d, q, 1, 0, e, 1); // e = Z*q;

	double qq = cblas_dnrm2(N, e, 1);
	qq = qq*qq;
	double qe = 0;
	for( int i=0; i<N; i++ ) qe += e[i];
	double ox = qe/qq;
	printf("ox=%f\n", ox);
	if( ox < 0.99*C && ox > 0.01*C ) {
		setmat(X, N, ox);
	} else if( ox > 0.99*C ) {
		setmat(X, N, 0.99*C);
	} else if( ox < 0.01*C ) {
		setmat(X, N, 0.01*C);
	}
	//writematrix("/Users/pwu/ownCloud/Projects/2019June_TensorSVM/X.csv", X, N, 1, 1);
	setmat(r, 3*N+1, 0);
	setmat(r_aff, 3*N+1, 0);
	setmat(S, N, 1.0);
	setmat(Xi, N, 1.0);

	double *D = (double*) malloc(sizeof(double)*N);
	int iter = 0;
	float *Zscaled = (float*) malloc(sizeof(float)*N*d*2); // dual use for single/double precision
	double *Zdscaled;
	double *M = (double*) malloc(sizeof(double)*d*d); // row-major
	float *Ms = (float*) malloc(sizeof(float)*d*d);
	double *work = (double*) malloc(sizeof(double)*5*N);
	lapack_int *ipiv = (lapack_int*) malloc(sizeof(lapack_int)*d);
	IPIV = ipiv;

	double *Md, *Zd; 
	const int NN = 100000;
	gpuErrchk( hipMalloc( &Md, sizeof(double)*d*d  ));
	gpuErrchk( hipMalloc( &Zd, sizeof(double)*NN*d ));
	double *Zt = (double*) malloc(sizeof(double)*NN*d); 
	double *Mt = (double*) malloc(sizeof(double)*d*d);

	for( iter=0; iter < MAX_MPC_ITER; iter++) {
		double mu = 0;
		//double cblas_sdot (const MKL_INT n, const double *x, const MKL_INT incx, const double *y, const MKL_INT incy);
		for( int i=0; i<N; i++) {
			mu += X[i]*S[i] + (C-X[i])*Xi[i];
		}
		mu = mu/(2*N);
		double *dx, *dy, *ds, *dxi;
		dx = r;
		dy = &r[N];
		ds = &r[N+1];
		dxi = &r[2*N+1];

		cblas_dgemv(CblasRowMajor, CblasTrans, N, d, 1.0, Z, d, X, 1, 0, q, 1); // q = Z'*X;
		cblas_dgemv(CblasRowMajor, CblasNoTrans, N, d, 1.0, Z, d, q, 1, 0, dx, 1); //dx = Z*q=Z*Z'*X
		for( int i=0; i<N; i++ ) dx[i] += -a[i]*y - S[i] + Xi[i] - 1.0;
		dy[0] = -cblas_ddot(N, a, 1, X, 1);
		for( int i=0; i<N; i++ ) ds[i] = -S[i]*X[i];
		for( int i=0; i<N; i++ ) dxi[i] = -Xi[i]*(C-X[i]);
		//writematrix("/Users/pwu/ownCloud/Projects/2019June_TensorSVM/r.csv", r, 3*N+1, 1, 1);

		for( int i=0; i<N; i++ ) {
			D[i] = S[i] / X[i] + Xi[i]/ (C-X[i]);
			// printf("D[%d]=%.3e ", i, D[i]);
		}
		//printf("\n");
		int imax = cblas_idamax(N, D, 1);
		int imin = cblas_idamin(N, D, 1);
		//printf("D is too ill-conditioned %.3e! Terminating.\n", D[imax]/D[imin]);

		double normdx = cblas_dnrm2(N, dx, 1);
		double normdy = fabs(dy[0]);
		//tmp = 0.5*X'*(Z*(Z'*X));
		cblas_dgemv(CblasRowMajor, CblasTrans, N, d, 1.0, Z, d, X, 1, 0, q, 1); // q = Z'*X;
		cblas_dgemv(CblasRowMajor, CblasNoTrans, N, d, 1.0, Z, d, q, 1, 0, e, 1); //dx = Z*q=Z*Z'*X
		double tmp = 0.5*cblas_ddot(N, X, 1, e, 1);
		double primalobj = tmp, dualobj=-tmp;
		for( int i=0; i<N; i++) {
			primalobj -= X[i];
			dualobj -= C*Xi[i];
		}
		if( iter%5 == 0)
			printf("iter %d, mu=%.3e, normdx=%.3e, normdy=%.3e max/min(D)=%.3e pobj=%.9e dobj=%.9e\n",
				iter, mu, normdx, normdy, D[imax]/D[imin], primalobj, dualobj);
		if( mu<1.e-7 && normdx<1.e-7 && normdy <1.e-7 ) {
			printf("Converged!\n");
			printf("iter %d, mu=%.3e, normdx=%.3e, normdy=%.3e max/min(D)=%.3e pobj=%.9e dobj=%.9e\n",
				iter, mu, normdx, normdy, D[imax]/D[imin], primalobj, dualobj);
			break;
		}
		if( D[imax]/D[imin] > 1e16 ) {
			printf("D is too ill-conditioned %.3e! Terminating.\n", D[imax]/D[imin]);
			printf("iter %d, mu=%.3e, normdx=%.3e, normdy=%.3e max/min(D)=%.3e pobj=%.9e dobj=%.9e\n",
				iter, mu, normdx, normdy, D[imax]/D[imin], primalobj, dualobj);
			break;
		}

		// scale the rows of Zscaled

		{ // change to double precision
			// printf("scaling Z N=%d C=%.3e\n", N, C);
			Zdscaled = (double*) Zscaled;
			memcpy(Zdscaled, Z, sizeof(double)*N*d);
			// writematrix("zscaled_pre.csv", Zdscaled, N, d, d);
			// writematrix("D.csv", D, N, 1, 1);
			for( int i=0; i<N; i++ ) {
				cblas_dscal(d, 1./sqrt(D[i]), &Zdscaled[i*d],  1);
			}
		}
		// writematrix("zscaled.csv", Zdscaled, N, d, d);
		clock_gettime(CLOCK_MONOTONIC, &start);	/* mark start time */
		{	// M = Zdscaled' * Zdscaled
            if ( 1.0*N*d*8 <= 4e9 ) {
                cblas_dsyrk(CblasRowMajor, CblasLower, CblasTrans, d, N, 1.0, Zdscaled, d, 0, M, d);
            } else {
                hipMemset( Md, 0, sizeof(double) * d * d );

                for(int i=0; i<N; i+=NN) {
                    int ib = min(NN, N-i); 

                    matcpy( ib, d, "ColMajor", Zt, ib, "RowMajor", &Zdscaled[i*d], d );
                    gpuErrchk( hipMemcpy( Zd, Zt, sizeof(double)*ib*d, hipMemcpyHostToDevice )); 
                    double done = 1.0; 
                    hipblasDsyrk(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_T, d, ib, &done,
                            Zd, ib, &done, Md, d);

                }
                gpuErrchk( hipMemcpy( Mt, Md, sizeof(double)*d*d, hipMemcpyDeviceToHost));
                matcpy(d,d,"RowMajor", M, d, "ColMajor", Mt, d);
            }
		}
		clock_gettime(CLOCK_MONOTONIC, &end);	/* mark the end time */
		diff = (end.tv_sec - start.tv_sec) + 1.0*(end.tv_nsec - start.tv_nsec)/BILLION;
		if (iter==0) printf("DSYRK elapsed time = %.3e seconds\n",  diff);

		



		for( int i=0; i<d; i++ ) M[i*d+i] += 1.0; // M = Z'*D^{-1}*Z + I
		//writematrix("/Users/pwu/ownCloud/Projects/2019June_TensorSVM/M.csv", M, d, d, d);
		// writematrix("Mfact.csv", M, d, d, d);
		int info = LAPACKE_dpotrf(LAPACK_ROW_MAJOR, 'L', d, M, d);
		
		if (info != 0) {
			printf("Cholesky fact info error %d; solver becomes unstable. Terminate.\n", info);
			return;
		}
		/* Experiment with LU instead of Chol; does not work and no warning.
		for( int i=0; i<d; i++ )
			for( int j=i; j<d; j++ )
				M[i*d+j] = M[j*d+i]; // symmetricalize M
		int info = LAPACKE_dgetrf(LAPACK_ROW_MAJOR, d, d, M, d, ipiv);
		if (info != 0) {
			printf("LU fact info error %d; solver becomes unstable. Terminate.\n", info);
			return;
		}*/

		// before = clock();
		clock_gettime( CLOCK_MONOTONIC, &start);
		NewtonStep(Z, D, M, C, a, X, S, Xi, r, work, d);
		clock_gettime( CLOCK_MONOTONIC, &end);
		diff = (end.tv_sec - start.tv_sec) + 1.0*(end.tv_nsec - start.tv_nsec)/BILLION;
		if (iter==0) printf("Newton elapsed time = %.3e seconds\n",  diff);
		// if(iter==0) printf("NewtonStep took %.3f seconds\n", 1.0*difference  / CLOCKS_PER_SEC);
		//writematrix("/Users/pwu/ownCloud/Projects/2019June_TensorSVM/r1.csv", r, 3*N+1, 1, 1);
		double alpha = 1;
		for( int i=0; i<N; i++ ) {
			if( dx[i] < 0 ) alpha = min( alpha, -X[i]/dx[i] );
			else if( dx[i] > 0 ) alpha = min(alpha, (C-X[i])/dx[i] );
			if( ds[i] < 0 ) alpha = min(alpha, -S[i]/ds[i]);
			if( dxi[i] < 0 ) alpha = min(alpha, -Xi[i]/dxi[i]);
		}
		//printf("alpha=%f\n", alpha);
		double mu_aff = 0;
		for( int i=0; i<N; i++ ) {
			mu_aff += (X[i] + alpha*dx[i]) * (S[i] + alpha*ds[i]) + (C-X[i]-alpha*dx[i])*(Xi[i]+alpha*dxi[i]) ;
		}
		mu_aff /= (2*N);
		//printf("mu_aff=%.3e\n", mu_aff);
		double sigma = (mu_aff/mu);
		sigma = sigma*sigma*sigma;

		memcpy(r_aff, r, sizeof(double)*(3*N+1));

		for( int i=0; i<N; i++) {
			ds[i] = sigma*mu - r_aff[i]*r_aff[i+N+1];
			dxi[i] = sigma*mu + r_aff[i]*r_aff[i+2*N+1];
			dx[i] = 0;
		}
		dy[0] = 0;
		NewtonStep(Z, D, M, C, a, X, S, Xi, r, work, d);
		for( int i=0; i<3*N+1; i++) {
			r[i] += r_aff[i];
		}
		alpha = 1;
		for( int i=0; i<N; i++ ) {
			if( dx[i] < 0 ) alpha = min( alpha, -X[i]/dx[i] );
			else if( dx[i] > 0 ) alpha = min(alpha, (C-X[i])/dx[i] );
			if( ds[i] < 0 ) alpha = min(alpha, -S[i]/ds[i]);
			if( dxi[i] < 0 ) alpha = min(alpha, -Xi[i]/dxi[i]);
		}
		alpha *= 0.99;
		//printf("corrector alpha %.3e\n", alpha);
		// update the variables:
		for( int i=0; i<N; i++ ) {
			X[i] += alpha*dx[i];
			S[i] += alpha*ds[i];
			Xi[i] += alpha*dxi[i];
		}
		y += alpha*dy[0];
	}
	//writematrix("/Users/pwu/ownCloud/Projects/2019June_TensorSVM/X.csv", X,N, 1, 1);
	free(S);  free(r); free(r_aff); free(work); free(q); free(e); free(D);
	free(Zscaled); free(M); free(Zt); free(Mt);
	hipFree(Md); hipFree(Zd); 

	hipblasDestroy(handle);
}


void setmat(double *mat, int n, double val)
{
	for(int i=0; i<n; i++) mat[i] = val;
}

// must supply 5*N elements in the work space
void NewtonStep(double *Z, double *D, double *M, double C, double *a, double *X, double *S, double *Xi, double *r, double *work, int d)
{
	double *r1 = r;
	double *r2 = &r[N];
	double *r3 = &r[N+1];
	double *r4 = &r[2*N+1];
	double *r5 = work;
	double r6;
	double *r7 = &work[N];
	double *b = &work[2*N];

	for( int i=0; i<N; i++ ) {
		r5[i] = r1[i] - r3[i]/X[i] + r4[i] / (C - X[i]);
		r7[i] = r5[i];
	}
    using namespace std::chrono;
    auto t1 = high_resolution_clock::now();
	SMWSolve(Z, D, M, r7, &work[3*N], d); // overwrites r7;
    auto t2 = high_resolution_clock::now();
    duration<double> time_span = duration_cast<duration<double>>(t2 - t1);
#ifdef DEBUG
    printf("SMWSolve in %.3f seconds.\n", time_span.count());
#endif
	r6 = r2[0] + cblas_ddot(N, a, 1, r7, 1);
	for( int i=0; i<N; i++ ) b[i] = a[i];
	SMWSolve(Z, D, M, b, &work[3*N], d);
	r2[0] = r6 / cblas_ddot(N, a, 1, b, 1);
	for( int i=0; i<N; i++ ) r1[i] = a[i]*r2[0] - r5[i];
	SMWSolve(Z, D, M, r1, &work[3*N], d);
	for( int i=0; i<N; i++ ) {
		r3[i] = (r3[i] - S[i]*r1[i]) / X[i];
		r4[i] = (r4[i] + Xi[i]*r1[i]) / (C-X[i]);
	}

}

void SMWSolve(double *Z, double *D, double *M, double *b, double *work, int d)
{
	double *c = work;
	for( int i=0; i<N; i++ ) {
		b[i] /= D[i];
		c[i] = b[i];
	}
	double *bb = &work[N];
	cblas_dgemv(CblasRowMajor, CblasTrans, N, d, 1.0, Z, d, b, 1, 0, bb, 1); // bb = Z'b

	LAPACKE_dpotrs(LAPACK_ROW_MAJOR, 'L', d, 1, M, d, bb, 1);
	//LAPACKE_dgetrs(LAPACK_ROW_MAJOR, 'N', d, 1, M, d, IPIV, bb, 1);
	cblas_dgemv(CblasRowMajor, CblasNoTrans, N, d, 1.0, Z, d, bb, 1, 0, b, 1); // b = Z*bb
	for( int i=0; i<N; i++ )
		b[i] = c[i] - b[i] / D[i];

}

// Low Rank Approximation of the Kernel Matrix:
// A = Z*Z', A is nxn, Z is nxk.
// A Z are row major.
// void LRA(double *A, int lda, double *Z, int ldz, int k)
// {
	

// }

// given a data matrix A, compute its kernel, multiplying an matrix B
// C = K(A) * B
void KerMatMul(double *A);




#if 0
void testKerMat(double *Z)
{
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("CUBLAS initialization failed\n");
		assert(0);
    }
	float diff;
	struct timespec start, end;
	int k = 1000;
	double *Zt = (double*) malloc( N*d*sizeof(double) );
	// row major -> col major on GPU
	for( int i=0; i<N; i++ )
		for( int j=0; j<d; j++)
			Zt[i+j*N] = Z[i*d+j];
	double *Q = (double *) malloc( N*k*sizeof(double) );
	for( int i=0; i<N; i++ ) {
		// Q[i] = gaussrand();
		// create an identity matrix
		for( int j=0; j<k; j++ ) {
			// if( j==i ) Q[i+j*N] = 1.0;
			// else Q[i+j*N] = 0;
			Q[i+j*N] = i+j;
		}
	}

	double *Qd, *Zd, *Yd, *Ld;
	gpuErrchk(hipMalloc( &Zd, sizeof(double)*N*d ));
	gpuErrchk(hipMalloc( &Qd, sizeof(double)*N*k ));
	gpuErrchk(hipMalloc( &Yd, sizeof(double)*N*k ));
	gpuErrchk(hipMalloc( &Ld, sizeof(double)*N ));
	printf("TEST: copying Z and D to device...");
	gpuErrchk(hipMemcpy( Zd, Zt, sizeof(double)*N*d, hipMemcpyHostToDevice ));
	gpuErrchk(hipMemcpy( Qd, Q, sizeof(double)*N*k, hipMemcpyHostToDevice ));
	gpuErrchk(hipMemcpy( Ld, LABELS, sizeof(double)*N, hipMemcpyHostToDevice ));
	printf("done.\n");
	clock_gettime(CLOCK_MONOTONIC, &start);	/* mark start time */
	printf("rbf_kermatmul: N=%d\n", N);
	rbf_kermatmul(Zd, N, Ld, Qd, N, Yd, N, N, k, handle);
	clock_gettime(CLOCK_MONOTONIC, &end);	/* mark the end time */
	diff = (end.tv_sec - start.tv_sec) + 1.0*(end.tv_nsec - start.tv_nsec)/BILLION;
	printf("elapsed time = %.3e seconds\n",  diff);

	double *Y = (double*) malloc( N*k*sizeof(double) );
	gpuErrchk(hipMemcpy( Y, Yd, sizeof(double)*N*k, hipMemcpyDeviceToHost ));
	if (N<=10 && k<=10) {
		printf("printing the kernel matrix %d\n", __LINE__);
		for( int i=0; i<N; i++ ) {
			for( int j=0; j<k; j++ ) {
				printf("%.6f ", Y[i+j*N]);
			}
			printf("\n");
		}
	} else {
		FILE *f = fopen("ker.csv", "w");
		for( int i=0; i<N; i++ ) {
			for( int j=0; j<k; j++ ) {
				fprintf(f, "%.6f", Y[i+j*N]);
				if( j<k-1 ) fprintf(f,",");
				else		fprintf(f, "\n");
			}

		}
		fclose(f);
	}
	hipblasDestroy(handle);
}
#endif

#ifdef CUDA
// matrix copy: A<-B

// ===========================================================================
// RBF kernel generation and low rank approximation on GPU
// !!!Note that All Matrices on GPUs are COLUMN MAJOR!!!
// ===========================================================================

// Z, U: row major
// K(Zd) \approx U*U'
// where U is n*k matrix. This is random projection based low rank
// matrix approxmation.
double LRA(float *Z, int ldz, double *U, int ldu, long n, long k)
{
	const int NN = 100000; // block size for out of core processing, 8192*12
	k = K;
	hipblasHandle_t handle;
	if (hipblasCreate(&handle) != HIPBLAS_STATUS_SUCCESS) {
        printf ("CUBLAS initialization failed\n");
        // return EXIT_FAILURE;
		assert(0);
	}
	hipsolverHandle_t cusolverH;
	statusH = hipsolverDnCreate(&cusolverH);
	if (statusH != HIPSOLVER_STATUS_SUCCESS) {
        printf ("CUSOLVER initialization failed\n");
        // return EXIT_FAILURE;
		assert(0);
	}		
	struct timespec start, end;
	double diff;

	// Q: n*k, row-major
	float *Q = (float*) malloc( sizeof(float)*n*k );
	int ldq = k; 
	for( int i=0; i<n; i++ ) {
		for( int j=0; j<k; j++ ) {
			Q[i*ldq+j] = gaussrand();
		}
	}
	// W: n*k, row-major
	float *W = (float*) malloc( N*k*sizeof(float) );
	int ldw = k; 

	int lwork = 0;
	float *d_work, *d_tau;
	int *d_info;
	float  *Cd;

	gpuErrchk( hipMalloc( &Cd, sizeof(float)*k*k) );
	gpuErrchk( hipMalloc( &d_tau, sizeof(float)*k ) );


	float *C = (float*) malloc( sizeof(float)*k*k );	// row-major


	for(int pr=0; pr<1; pr++) {


		//rbf_kermatmul(Zd, N, Yd, Qd, N, Wd, N, N, k, handle);
		// W = K*Q, on CPU. W,K,Q are all row-major
		//int d = d; 
		auto RBF_KERMATMUL = [k,Z,ldz, handle](float *Q, int ldq, float *W, int ldw, int d)
		{ 
			printf(" in %d chunks ", (N+NN-1)/NN); 
			float *Z1t = (float*) malloc( sizeof(float) * NN * d );
			float *Z2t = (float*) malloc( sizeof(float) * NN * d );
			float *Qt  = (float*) malloc( sizeof(float) * NN * k );
			float *Wt  = (float*) malloc( sizeof(float) * NN * k );

			float *Zd1, *Zd2, *Yd1, *Yd2, *Qd, *Wd; 
			gpuErrchk( hipMalloc( &Zd1, sizeof(float) * NN * d ));
			gpuErrchk( hipMalloc( &Zd2, sizeof(float) * NN * d ));
			gpuErrchk( hipMalloc( &Yd1, sizeof(float) * NN     ));
			gpuErrchk( hipMalloc( &Yd2, sizeof(float) * NN     ));
			gpuErrchk( hipMalloc( &Qd,  sizeof(float) * NN * k ));
			gpuErrchk( hipMalloc( &Wd,  sizeof(float) * NN * k ));

			for (int i=0; i<N; i+=NN) { 
				int rn = min(NN, N-i); 
				gpuErrchk( hipMemset(Wd, 0, sizeof(float)*NN*k) ); //Wd = 0; 
				for (int j=0; j<N; j+=NN) { 
					int cn = min(NN, N-j); 
					
					matcpy(rn, d, "ColMajor", Z1t, rn, "RowMajor", &Z[i*ldz], ldz );
					matcpy(cn, d, "ColMajor", Z2t, cn, "RowMajor", &Z[j*ldz], ldz );
					matcpy(cn, k, "ColMajor", Qt,  cn, "RowMajor", &Q[j*ldq], ldq );

					gpuErrchk( hipMemcpy( Zd1, Z1t, sizeof(float)*rn*d, hipMemcpyHostToDevice) );
					gpuErrchk( hipMemcpy( Zd2, Z2t, sizeof(float)*cn*d, hipMemcpyHostToDevice) );
					gpuErrchk( hipMemcpy( Yd1, &LABELS[i], sizeof(float) * rn, hipMemcpyHostToDevice) );
					gpuErrchk( hipMemcpy( Yd2, &LABELS[j], sizeof(float) * cn, hipMemcpyHostToDevice) );
					gpuErrchk( hipMemcpy( Qd, Qt, sizeof(float)*cn*k, hipMemcpyHostToDevice) );

					// Wd += K[i,j]*Q[j] 
					//printf("RBFKER: d=%d k=%d (i,j)=(%d,%d) (rn,cn)=(%d,%d) \n",
					//	               d,   k, i, j,         rn, cn); 
					rbf_kermatmul(Zd1, rn, Zd2, cn, Yd1, Yd2, Qd, cn, Wd, rn, 
					   /*sizes*/  rn, cn, k, handle);
										
				}
				gpuErrchk( hipMemcpy( Wt, Wd, sizeof(float) * rn * k, hipMemcpyDeviceToHost ) ); 
				matcpy(rn, k, "RowMajor", &W[i*k], k, "ColMajor", Wt, rn); 
			}

			free(Z1t); free(Z2t); free(Qt); free(Wt); 
			hipFree(Zd1); hipFree(Zd2); hipFree(Yd1); hipFree(Yd2); hipFree(Qd); hipFree(Wd);
		};
		
		printf("rbf_kermatmul1 ");
		START_TIMER
		RBF_KERMATMUL(Q, ldq, W, ldw, d); 
		END_TIMER

			// W = Ortho(W)
			//void ortho(float *, int, int);
			//auto ortho = [](float *W, int n, int k){};
		printf("Ortho ");
		START_TIMER
		ortho(W, n, k); // W is row-major
		END_TIMER
		


		// C= W'*K*W -> Q = K*W; C=W'*Q
		/* Q=K*W */
		//rbf_kermatmul(Zd, N, Yd, Wd, N, Qd, N, N, k, handle);
		printf("rbf_kermatmul2 ");
		START_TIMER
		RBF_KERMATMUL(W, ldw, Q, ldq, d); 
		END_TIMER


		//hipMemcpy(Qd, Wd, sizeof(double)*n*k, hipMemcpyDeviceToDevice);
		// hipFree(d_work);
	}
	/* C=W'*Q */
	//hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, k, k, N, &done, Wd, N, Qd, N, &dzero,
	//			Cd, k);
	printf("Compute C ");
START_TIMER
#ifdef C_FP32
	{
		// use Cd; col-major
		float *Wd, *Qd; 
		gpuErrchk( hipMalloc( &Wd, sizeof(float) * NN * k )); 
		gpuErrchk( hipMalloc( &Qd, sizeof(float) * NN * k )); 
		float *Wt = (float*) malloc( sizeof(float) * NN * k );  
		float *Qt = (float*) malloc( sizeof(float) * NN * k ); 

		hipMemset( Cd, 0, sizeof(float)*k*k ); // Cd=0
		
		for(int i=0; i<N; i+=NN) {
			int ib = min(NN, N-i); 
			matcpy( ib, k, "ColMajor", Wt, ib, "RowMajor", &W[i*ldw], ldw);
			matcpy( ib, k, "ColMajor", Qt, ib, "RowMajor", &Q[i*ldq], ldq); 
			gpuErrchk( hipMemcpy( Wd, Wt, sizeof(float)*ib*k, hipMemcpyHostToDevice));
			gpuErrchk( hipMemcpy( Qd, Qt, sizeof(float)*ib*k, hipMemcpyHostToDevice));
			hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, k, k, ib, &done,
						Wd, ib, Qd, ib, &dzero/* should be $done?? */, Cd, k); 
		}
		// need to make C symmetric! 
		hipFree(Wd); hipFree(Qd); 
		free(Wt); free(Qt); 
		gpuErrchk(hipMemcpy( C, Cd, sizeof(float)*k*k, hipMemcpyDeviceToHost ));
	}
#else	// C FP64, C = W'*Q
	{
		// use Cd; col-major
		double *Wd, *Qd; 
		gpuErrchk( hipMalloc( &Wd, sizeof(double) * NN * k )); 
		gpuErrchk( hipMalloc( &Qd, sizeof(double) * NN * k )); 
		double *Wt = (double*) malloc( sizeof(double) * NN * k );  
		double *Qt = (double*) malloc( sizeof(double) * NN * k ); 

		double *Cd64; 
		hipMalloc( &Cd64, sizeof(double)*k*k ); 
		hipMemset( Cd64, 0, sizeof(double)*k*k ); // Cd=0
		
		printf(" in %d chunks", (N+NN-1)/NN); 
		for(int i=0; i<N; i+=NN) {
			int ib = min(NN, N-i); 
			matcpy( ib, k, "ColMajor", Wt, ib, "RowMajor", &W[i*ldw], ldw);
			matcpy( ib, k, "ColMajor", Qt, ib, "RowMajor", &Q[i*ldq], ldq); 
			gpuErrchk( hipMemcpy( Wd, Wt, sizeof(double)*ib*k, hipMemcpyHostToDevice));
			gpuErrchk( hipMemcpy( Qd, Qt, sizeof(double)*ib*k, hipMemcpyHostToDevice));
			double done = 1; 
			hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, k, k, ib, &done,
						Wd, ib, Qd, ib, &done, Cd64, k); 
		}
		hipFree(Wd); hipFree(Qd); 
		free(Wt); free(Qt); 
		double *CC = new double[k*k];
		gpuErrchk(hipMemcpy( CC, Cd64, sizeof(double)*k*k, hipMemcpyDeviceToHost ));
		matcpy( k, k, "ColMajor", C, k, "ColMajor", CC, k); 
		// important step: make C symmetric. 
		for (int i=0; i<k; i++) {
			for (int j=0; j<i; j++) {
				C[i+j*k] = 0.5*(C[i+j*k] + C[j+i*k]);
			}
		}
		gpuErrchk(hipMemcpy( Cd, C, sizeof(float)*k*k, hipMemcpyHostToDevice)); 
		free(CC); 
		hipFree(Cd64);
	}
#endif
	printf (" done "); 
END_TIMER
	// is C symmetric? 
	
#ifdef DEBUG
	FILE *f3 = fopen("C.csv","w");
	for( int i=0; i<k; i++ ){
		for( int j=0; j<k; j++ ){
			fprintf(f3, "%.6f", C[i+j*k]);
			if( j<k-1 ) fprintf(f3,",");
			else		fprintf(f3, "\n");
		}
	}
	fclose(f3);
#endif

// C_EIG: C=L*L' through eigen analysis, otherwise with Cholesky. 
#define C_EIG
#ifdef C_EIG
START_TIMER
	{
		printf("eigen decomposition of C..."); 
	// spectral analysis of C (therefore the low rank apprixmation)
		double *CC = (double*) malloc( sizeof(double)*k*k );
		double *w = (double*) malloc( sizeof(double)*k ); // ews in ascending order
		float *X = new float[k*k]; 

		//memcpy(CC, C, sizeof(double)*k*k);
		matcpy( k, k, "ColMajor", CC, k, "ColMajor", C, k);
		int info = LAPACKE_dsyevd( LAPACK_COL_MAJOR, 'V', 'L', k, CC, k, w);
		if (info != 0) {
			printf("Error: DSYEVD info=%d", info); 
		}
		printf("[LRA]: C: largest ew=%.3e, smallest ew=%.3e\n", w[k-1], w[0]);
		// eigenvalues in w in ascending order. 
		int i, realk=k; 
		// C = X*X'
		for (i=0; i<k; i++) {
			double s; 
			if (w[i] > 0 ) {
				s = sqrt(w[i]); 
			} else {
				s = 0;
				realk--;
			}
			for (int j=0; j<k; j++) {
				X[j + i*k] = s * CC[j + i*k]; 
			}
		}
		printf("real rank is %d", realk); 

		printf(" U=W*X  ");
		{
			float *Wt = (float*) malloc( sizeof(float) * NN * k );
			float *Ut = (float*) malloc( sizeof(float) * NN * k );
			float *Wd, *Ud, *Xd; 
			gpuErrchk( hipMalloc( &Wd, sizeof(float) * NN * k ) );
			gpuErrchk( hipMalloc( &Ud, sizeof(float) * NN * k ) );

			gpuErrchk( hipMalloc( &Xd, sizeof(float) * k * k ) );
			gpuErrchk( hipMemcpy( Xd, X, sizeof(float) * k * k, hipMemcpyHostToDevice));

			//gpuErrchk( hipMemset( Ud, 0, sizeof(float)*NN*k ) ); // Ud = 0. 
			printf(" in %d chunks", (N+NN-1)/NN); 
			for (int i=0; i<N; i+=NN) {
				int ib = min(NN, N-i); 
				matcpy(ib, k, "ColMajor", Wt, ib, "RowMajor", &W[i*ldw], ldw);
				gpuErrchk( hipMemcpy( Wd, Wt, sizeof(float) * ib * k, hipMemcpyHostToDevice));
				float sone = 1.0, szero = 0.0; 
				hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
							ib, k, k, 
							&sone, Wd, ib, Xd, k, &szero, Ud, ib); 
				gpuErrchk( hipMemcpy( Ut, Ud, sizeof(float) * ib * k, hipMemcpyDeviceToHost));
				matcpy(ib, k, "RowMajor", &U[i*ldu], ldu, "ColMajor", Ut, ib); 
			}
			free(Wt); free(Ut);
			gpuErrchk(hipFree(Wd)); 
			gpuErrchk(hipFree(Ud));
			gpuErrchk(hipFree(Xd));
		}
		printf(" done. "); 

		double l1 = w[k-1];
		free(CC);
		free(w);
		delete[] X; 
	}
END_TIMER
#else

	printf("C Cholesky factorize...");
START_TIMER
#ifdef C_GPU
	{				
		printf(" on GPU ")
		// C=L*L' on GPU
		int info; 
		gpuErrchk( hipMalloc( &d_info, sizeof(int)));
		statusH = hipsolverDnSpotrf_bufferSize(cusolverH, HIPBLAS_FILL_MODE_LOWER, k, Cd,k,  &lwork );
		assert(statusH == HIPSOLVER_STATUS_SUCCESS);
		gpuErrchk( hipMalloc( &d_work, sizeof(float)*lwork ) );
		statusH = hipsolverDnSpotrf(cusolverH, HIPBLAS_FILL_MODE_LOWER, k, Cd ,k, d_work, lwork, d_info );
		assert(statusH == HIPSOLVER_STATUS_SUCCESS);
		hipFree( d_work );
		gpuErrchk( hipMemcpy( &info, d_info, sizeof(int), hipMemcpyDeviceToHost) );
		if (info!=0) {
			printf("Cholesky fail; info=%d\n", info);
			exit(1); 
		}
		hipFree( d_info);
	}
#else 
	{  // C=L*L' on CPU
		printf(" on CPU "); 
		float *CC = new float[k*k]; 
		matcpy( k, k, "ColMajor", CC, k, "ColMajor", C, k);

		int info = LAPACKE_spotrf( LAPACK_COL_MAJOR, 'L', k, C, k ); 
		if (info !=0 ) {
			printf("CPU Cholesky fail; info =%d, attempting to guess numerical rank...\n", info); 

			int *ipiv = new int[k];
			int rank; 
			int info = LAPACKE_spstrf( LAPACK_COL_MAJOR, 'L', k, CC, k, ipiv, &rank, 1e-7 );
			printf("CPU Pivoted Cholesky: info=%d rank=%d (retry setting -k=%d or less)\n", info, rank, rank); 
			exit(1);

			delete[] ipiv; 

		}
		gpuErrchk( hipMemcpy( Cd, C, sizeof(float)*k*k, hipMemcpyHostToDevice ) );
		delete[] CC; 

	}

	printf("done\n");
END_TIMER

	// U = W*L; L is already stored in Cd on GPU. 
	printf(" U=W*L  \n");
	{
		float *Wt = (float*) malloc( sizeof(float) * NN * k );
		float *Ut = (float*) malloc( sizeof(float) * NN * k );
		float *Wd, *Ud; 
		gpuErrchk( hipMalloc( &Wd, sizeof(float) * NN * k ) );
		gpuErrchk( hipMalloc( &Ud, sizeof(float) * NN * k ) );
		hipMemset( Ud, 0, sizeof(float)*NN*k );
		for (int i=0; i<N; i+=NN) {
			int ib = min(NN, N-i); 
			matcpy(ib, k, "ColMajor", Wt, ib, "RowMajor", &W[i*ldw], ldw);
			gpuErrchk( hipMemcpy( Wd, Wt, sizeof(float) * ib * k, hipMemcpyHostToDevice));
			float sone = 1.0; 
			hipblasStrmm(handle, HIPBLAS_SIDE_RIGHT, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N,
						HIPBLAS_DIAG_NON_UNIT,
						ib, k, &sone, Cd, k, Wd, ib, Ud, ib); 
			gpuErrchk( hipMemcpy( Ut, Ud, sizeof(float) * ib * k, hipMemcpyDeviceToHost));
			matcpy(ib, k, "RowMajor", U, ldu, "ColMajor", Ut, ib); 
		}
		free(Wt); free(Ut);
		gpuErrchk(hipFree(Wd)); 
		gpuErrchk(hipFree(Ud));
	}
	printf(" done. \n"); 
#endif	
#endif

	if (flag_analysis) {
		// WARNING: THis piece of code does not work with OOC. 
		void rbf_fnorm_res(double *Zd, int ldz, double *Yd, double *U, int ldu, int n, int k,
						   double *fnorm, double *fnorm_res, hipblasHandle_t handle);

		double *Zd, *Ud, *Yd;
		double *Ut = new double[n*k]; 
		double *Yt = new double[n];
		double *Zt = new double[n*d];
		START_TIMER

		matcpy( n, d, "ColMajor", Zt, n, "RowMajor", Z, ldz );
		matcpy( n, k, "ColMajor", Ut, n, "RowMajor", U, ldu );	
		matcpy( n, 1, "ColMajor", Yt, n, "RowMajor", LABELS, 1 );	

		gpuErrchk( hipMalloc( &Zd, sizeof(double)*n*d ));
		gpuErrchk( hipMalloc( &Ud, sizeof(double)*n*k ));
		gpuErrchk( hipMalloc( &Yd, sizeof(double)*n ));	
		gpuErrchk( hipMemcpy( Zd, Zt, sizeof(double)*n*d, hipMemcpyHostToDevice ));			
		gpuErrchk( hipMemcpy( Ud, Ut, sizeof(double)*n*k, hipMemcpyHostToDevice ));			
		gpuErrchk( hipMemcpy( Yd, Yt, sizeof(double)*n*1, hipMemcpyHostToDevice ));	

		double knorm = 0, kunorm = 0;						
		rbf_fnorm_res(Zd, N, Yd, Ud, n, n, k, &knorm,&kunorm,  handle);
		printf("fnorm(K)=%.3e fnorm(K-U*U=%.3e')\n", knorm, kunorm);
		printf("rbf_fnorm_res ");
		END_TIMER

		

		delete[] Ut; delete[] Yt; delete[] Zt; 
		hipFree(Ud); hipFree(Yd); hipFree(Zd); 
	}


	hipFree(Cd);
	hipFree(d_tau);
	free(Q);
	free(W);
	free(C);

	hipblasDestroy(handle);
	hipsolverDnDestroy(cusolverH);
	//return l1;
	return 0;
}
		

template <typename T>
struct square
{
	__host__ __device__
	T operator()(const T& x) const { 
		return x * x;
	}
};
// residual f-norm of the LRA result:
// fnorm(K - U*U')/fnorm(K);

void rbf_fnorm_res(double *Zd, int ldz, double *Yd, double *Ud, int ldu, int n, int k,
				   double *fnorm, double *fnorm_res, hipblasHandle_t handle)
{
	// ops for Thrust transform-reduction
	square<double>        unary_op;
	thrust::plus<double> binary_op;

	*fnorm = 0;
	// determine a block size
	int B = 8192;
	double *buf;
	double *XIJ, *XI, *XJ;
	gpuErrchk(hipMalloc( &buf, B*B*sizeof(double) ));
	gpuErrchk(hipMalloc( &XIJ, B*B*sizeof(double) ));
	gpuErrchk(hipMalloc( &XI, B*sizeof(double) ));
	gpuErrchk(hipMalloc( &XJ, B*sizeof(double) ));



	double acc1 = 0;
	double acc2 = 0;
	
	double done = 1;
	double dzero = 0;
	double none=-1; 
	for (int i=0; i<n; i+=B) {
		int ib = min(B, n-i);
		for (int j=0; j<n; j+=B) {
			int jb = min(B, n-j);
			// printf("i=%d j=%d", i,j);
			// step 1: populate XI, XJ, XIJ
			vecnorm<<<(B+63)/64, 64>>>(&Zd[i], ldz, XI, ib, d);
			gpuErrchk( hipPeekAtLastError() );
			gpuErrchk( hipDeviceSynchronize() );
			vecnorm<<<(B+63)/64, 64>>>(&Zd[j], ldz, XJ, jb, d);
			gpuErrchk( hipPeekAtLastError() );
			gpuErrchk( hipDeviceSynchronize() );
			// XIJ is column major!!
			// printf("ib=%d jb=%d d=%d ldz=%d\n", ib, jb, d, ldz);
			stat = hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, ib, jb, d,
							   &done, &Zd[i], ldz,
							   &Zd[j], ldz, &dzero,
							   XIJ, ib);
			if (stat != HIPBLAS_STATUS_SUCCESS) 
				printf ("hipblasDgemm failed %s\n", __LINE__);
			gpuErrchk( hipPeekAtLastError() );
			gpuErrchk( hipDeviceSynchronize() );
			dim3 threadsPerBlock(32,32);
			dim3 numBlocks( (ib+threadsPerBlock.x-1)/threadsPerBlock.x,
							(jb+threadsPerBlock.y-1)/threadsPerBlock.y );
			// printf("ib=%d, jb=%d, B=%d, TPB.(x,y)=(%d,%d), B.(x,y)=(%d,%d)\n",
			// 	   ib, jb, B, threadsPerBlock.x, threadsPerBlock.y,
			// 	   numBlocks.x, numBlocks.y);
			rbf_kergen<<<numBlocks, threadsPerBlock>>>( ib, jb, buf, ib, XI, XJ, XIJ, ib, g, &Yd[i], &Yd[j]);
			gpuErrchk( hipPeekAtLastError() );
			gpuErrchk( hipDeviceSynchronize() );

			// trying Thrust!

			thrust::device_ptr<double> buf_ptr(buf);
			double init = 0;
			// compute norm
			acc1 += thrust::transform_reduce(buf_ptr, buf_ptr+ib*jb, unary_op, init, binary_op);

			// buf -= Ui * Uj'
			hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, ib, jb, k, &none, &Ud[i], ldu, &Ud[j], ldu, &done,
						buf, ib);
			acc2 += thrust::transform_reduce(buf_ptr, buf_ptr+ib*jb, unary_op, init, binary_op);
		}
	}
	*fnorm = sqrt( acc1 );
	*fnorm_res = sqrt( acc2 );
		
	hipFree(buf);
	hipFree(XI);
	hipFree(XJ);
	hipFree(XIJ);
	hipDeviceSynchronize();	

}


// compute kernel matrix-matrix multiplication:
// B += Y*K(Z)*Y'*A;
// All matrices are col-major
// kernel is m*n, B is m * k;
// A is of size n*k. k could be 1, which gives matrix-vector multiplication.
// the suffix -d suggests the pointer points to device memory space.
void rbf_kermatmul(float *Zd1, int ldz1, float *Zd2, int ldz2, float *Yd1, float *Yd2,
	float *Ad, int lda, float *Bd, int ldb,  int m, int n, int k,
	hipblasHandle_t handle)
{


	// determine a block size
	const int B = 8192;
	float *buf; // stores the temporary kernel matrix block of size B*B
	float *XIJ, *XI, *XJ;
	gpuErrchk(hipMalloc( &buf, B*B*sizeof(float) ));
	gpuErrchk(hipMalloc( &XIJ, B*B*sizeof(float) )); // XIJ[i,j] = XI[i]'* XJ[j];
	gpuErrchk(hipMalloc( &XI, B*sizeof(float) ));
	gpuErrchk(hipMalloc( &XJ, B*sizeof(float) ));

	float sone = 1;
	float szero = 0;
	for (int i=0; i<m; i+=B) {
		int ib = min(B, m-i);
		for (int j=0; j<n; j+=B) {
			int jb = min(B, n-j);
			// step 1: populate XI, XJ, XIJ
			//printf("vecnorm: (i,j)=(%d,%d) (ib,jb)=(%d,%d) (ldz1,ldz2)=(%d,%d), d=%\n", i, j, ib, jb, ldz1, ldz2, d);
			vecnorm<<<(B+63)/64, 64>>>(&Zd1[i], ldz1, XI, ib, d);
			vecnorm<<<(B+63)/64, 64>>>(&Zd2[j], ldz2, XJ, jb, d);
			gpuErrchk( hipPeekAtLastError() );
			//gpuErrchk( hipDeviceSynchronize() );
			// XIJ is column major!!
			if (flag_tensorcore && d >= 256) {
				stat = hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, ib, jb, d,
								   &sone, &Zd1[i], HIP_R_32F,ldz1,
								   &Zd2[j], HIP_R_32F,ldz2, &szero,
								   XIJ, HIP_R_32F, ib,
								   HIP_R_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP);	
			} else {
				stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, ib, jb, d,
								   &sone, &Zd1[i], ldz1,
								   &Zd2[j], ldz2, &szero,
								   XIJ, ib);		
	
			}
			if (stat != HIPBLAS_STATUS_SUCCESS) 
				printf ("hipblasSgemm failed %s\n", __LINE__);
			gpuErrchk( hipPeekAtLastError() );
			//gpuErrchk( hipDeviceSynchronize() );
			dim3 threadsPerBlock(32,32);
			dim3 numBlocks( (ib+threadsPerBlock.x-1)/threadsPerBlock.x,
							(jb+threadsPerBlock.y-1)/threadsPerBlock.y );
			// printf("ib=%d, jb=%d, B=%d, TPB.(x,y)=(%d,%d), B.(x,y)=(%d,%d)\n",
			// 	   ib, jb, B, threadsPerBlock.x, threadsPerBlock.y,
			// 	   numBlocks.x, numBlocks.y);
			rbf_kergen<<<numBlocks, threadsPerBlock>>>( ib, jb, buf, B, XI, XJ, XIJ, ib, g, &Yd1[i], &Yd2[j]);
			gpuErrchk( hipPeekAtLastError() );
			//gpuErrchk( hipDeviceSynchronize() );
			if (k>1) {
			// this works for both k=1 or k>1.
				if (!flag_tensorcore || k < 256) {
					hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, ib, k, jb,
								&sone, buf, B, &Ad[j], lda,
								&sone, &Bd[i], ldb);
				}
				else {
					hipblasGemmEx( handle, HIPBLAS_OP_N, HIPBLAS_OP_N, ib, k, jb, 
								  &sone, buf, HIP_R_32F, B, &Ad[j], HIP_R_32F, lda,
					 			  &sone, &Bd[i], HIP_R_32F, ldb, 
					 			  HIP_R_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP);
				}
			} else if (k==1) {
			// 	// printf("Unimplemented! %s\n", __LINE__);
			// 	// exit(-1);
				hipblasSgemv(handle, HIPBLAS_OP_N, ib, jb, &sone,
							buf, B, &Ad[j], 1, &sone, &Bd[i], 1);
			}
		}
	}

		
	hipFree(buf);
	hipFree(XI);
	hipFree(XJ);
	hipFree(XIJ);
	hipDeviceSynchronize();
}

template<typename FloatType>
__global__ void
vecnorm(FloatType *Zd, int ldz, FloatType *ZI, int m, int k)
{
	int i=blockIdx.x*blockDim.x + threadIdx.x;
	if( i<m ) {
		FloatType sum = 0;
#pragma unroll (4)
		for( int j=0; j<k; j++ )
			sum += Zd[i+j*ldz]*Zd[i+j*ldz];
		ZI[i] = sum;
	}
}
__global__ void
fnorm( int m, int n, double *buf, int B, double *XI, double *XJ, double *XIJ, int ldxij, double gamma, double *YI,
	   double *YJ, double *acc)
{
	int i=blockIdx.x*blockDim.x + threadIdx.x;
	int j=blockIdx.y*blockDim.y + threadIdx.y;

	if (i<m && j<n) 
		// buf[i+j*ldb] = YI[i]*YJ[j]*__expf(-gamma*(XI[i] + XJ[j] - 2*XIJ[i+j*ldxij]));
		acc += 0;
}
// CUDA kernel to generate the matrix, block by block.
// the result will be store in buf, column major, m*n matrix,
// with LDA m.
// XIJ/buf are column major.
// could be improved by assigning more work to each thread.
template<typename FloatType, typename GammaType>
__global__
void rbf_kergen( int m, int n, FloatType *buf, int ldb,
				 FloatType *XI, FloatType *XJ, FloatType *XIJ, int ldxij,
				 GammaType gamma, FloatType *YI, FloatType *YJ)
{
	int i=blockIdx.x*blockDim.x + threadIdx.x;
	int j=blockIdx.y*blockDim.y + threadIdx.y;

	if (i<m && j<n) {
		buf[i+j*ldb] = YI[i]*YJ[j]*__expf(-gamma*(XI[i] + XJ[j] - 2*XIJ[i+j*ldxij]));
		// printf("[i,j]=[%d,%d], buf[]=%.4f, XI[]=%.4f, XJ[]=%.4f, XIJ[]=%.4f\n", i, j, buf[i+j*ldb],
		// 	   XI[i], XJ[j], XIJ[i+j*ldxij]);
	}
}

#endif
// end of CUDA

// ===========================================================================
// Auxillary functions.
// ===========================================================================

float gaussrand()
{
    static float V1, V2, S;
    static int phase = 0;
    float X;

    if(phase == 0) {
        do {
            float U1 = (float)rand() / RAND_MAX;
            float U2 = (float)rand() / RAND_MAX;

            V1 = 2 * U1 - 1;
            V2 = 2 * U2 - 1;
            S = V1 * V1 + V2 * V2;
        } while(S >= 1 || S == 0);

        X = V1 * sqrt(-2 * log(S) / S);
    } else
        X = V2 * sqrt(-2 * log(S) / S);

    phase = 1 - phase;
    return X;
}
